#include <groot.h>
#include <string>
#include <filesystem>

using namespace groot;

// Configuration for the converter application
struct ConverterConfig {
    std::string input_file;
    std::string output_file;
    std::string output_original_file;
    ReorderAlgo reorder = ReorderAlgo::Groot;
};

// Generate automatic filenames based on input file
void generate_automatic_filenames(ConverterConfig& config) {
    if (config.input_file.empty()) {
        return;
    }

    // Extract base name without extension
    std::string base_name;
    size_t last_dot = config.input_file.find_last_of(".");
    size_t last_slash = config.input_file.find_last_of("/\\");

    if (last_slash == std::string::npos) {
        // No path separator found
        base_name = config.input_file.substr(0, last_dot);
    } else {
        // Extract filename part after the path separator
        base_name = config.input_file.substr(last_slash + 1, last_dot - last_slash - 1);
        // Add directory path back
        base_name = config.input_file.substr(0, last_slash + 1) + base_name;
    }

    // If output files not specified, set default names
    if (config.output_original_file.empty()) {
        config.output_original_file = base_name + ".mtx";
    }

    if (config.output_file.empty()) {
        config.output_file = base_name + "_groot.mtx";
    }
}

// Parse command-line arguments for the converter
ConverterConfig converter_program_options(int argc, char* argv[])
{
    ConverterConfig config;
    int opt;
    if (argc == 1) {
        printf("Usage: %s ... \n%s", argv[0],
            "              [-i input_file] Input CSR file\n"
            "              [-o output_file] (Optional) Output MTX file after reordering\n"
            "              [-p output_original_file] (Optional) Output MTX file before reordering\n"
            "              [-r reorder_algorithm (0: none, 1: groot)]\n\n"
            "If output files are not specified, they will be derived from the input filename:\n"
            "For input 'file.csr', outputs will be 'file.mtx' and 'file_reordered.mtx'\n");
        std::exit(EXIT_FAILURE);
    }
    while ((opt = getopt(argc, argv, "i:o:p:r:")) != -1) {
        switch (opt) {
            case 'i':
                config.input_file = optarg;
                break;
            case 'o':
                config.output_file = optarg;
                break;
            case 'p':
                config.output_original_file = optarg;
                break;
            case 'r':
                config.reorder = static_cast<ReorderAlgo>(std::stoi(optarg));
                break;
            default:
                printf("Usage: %s ... \n%s", argv[0],
                    "              [-i input_file] Input CSR file\n"
                    "              [-o output_file] (Optional) Output MTX file after reordering\n"
                    "              [-p output_original_file] (Optional) Output MTX file before reordering\n"
                    "              [-r reorder_algorithm (0: none, 1: groot)]\n");
                exit(EXIT_FAILURE);
        }
    }

    // Generate automatic filenames if not specified
    generate_automatic_filenames(config);

    printf("--------experimental setting--------\n");
    if (!config.input_file.empty()) {
        printf("input path: %s\n", config.input_file.c_str());
    }
    printf("output original path: %s\n", config.output_original_file.c_str());
    printf("reorder algorithm: %s\n", reorder_algo_to_string(config.reorder));
    printf("output reordered path: %s\n", config.output_file.c_str());

    return config;
}

int main(int argc, char** argv)
{
    hipSetDevice(0);

    CsrMatrix<int, float, device_memory> A_csr;

    ConverterConfig config = converter_program_options(argc, argv);

    // Read the input matrix
    read_matrix_file(A_csr, config.input_file);

    // Write the original matrix to a MTX file
    write_matrix_file(A_csr, config.output_original_file);

    // Reorder the graph
    reorder_graph(config, A_csr);

    // Write the reordered matrix to a MTX file
    write_matrix_file(A_csr, config.output_file);

    return 0;
}