#include <groot.h>

using namespace groot;

int main(int argc, char** argv)
{
    hipSetDevice(0);

    CsrMatrix<int, float, device_memory> A_csr;

    Config config = program_options(argc, argv);

    read_matrix_file(A_csr, config.input_file);

    reorder_graph(config, A_csr);

    write_matrix_file(A_csr, config.output_file);

    return 0;
}
